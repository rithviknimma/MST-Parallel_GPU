#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <time.h>
// #include <thrust/device_ptr.h>
// #include <thrust/device_vector.h>
// #include <thrust/sort.h>
// #include <thrust/scan.h>

#define THREADSPERBLOCK 512

// graph
struct edge{
	int v;
	int u;
	int weight;
};

struct graph{
	int num_edges;
	int num_vertices;
	struct edge* edges;
};

// bipartite graph
struct b_vertex_a{
	int v;
	int small_edge; // don't know what this is for
};

struct b_vertex_b{
	int e; // edge number
};

struct b_edge{
	int v;
	int u;
	int cv;
	int weight;
};

struct b_graph{
	int num_vertex_a;
	int num_vertex_b;
	int num_bipartite_edges;
	struct b_vertex_a* vertices_a;
	struct b_vertex_b* vertices_b; 
	struct b_edge* edges;
};

// strut
struct strut_edge{
    int v;
    int u; // edge number index
    int cv; // correspondent vertex
};

struct strut_u_vertex{ 
    int degree; // degree in struts
    int v1;
    int v2;
    int weight;
};

struct strut{
    int num_v; // num of bipartite vertices
    int num_u; // num of u vertices adjacent to strut edge
    int num_strut_edges; // same number as num_v
    struct strut_edge* edges; 
    struct strut_u_vertex* vertices_u; // u vertices - 0 value indicates not in strut, value > 0 indicates how many strut edges it is connected to
};

void get_graph(struct graph* og_graph, char* input);
__global__ void get_bipartite_graph(int num_edges, int num_vertices, struct edge* graphEdges, struct b_vertex_a* vetices_a, struct b_vertex_b* vetices_b, struct b_edge* bg_graphEdges) ;


__global__ void get_smallest_edges(int bp_num_edges, int num_smallest_edges, struct b_edge* bg_graphEdges, int* smallest_weights, int* smallest_edges);
__global__ void mst_edges_init(int og_num_edges, bool *mst_edges);
__global__ void get_mst_edges(int num_smallest_edges, int* smallest_edges, struct b_edge* bg_graphEdges, bool *mst_edges);
__global__ void get_num_mst(int og_num_edges, bool *mst_edges, int* num_mst);

// strut stuff
__global__ void get_strut_edges(int bg_num_vertices, int* smallest_edges, struct b_edge* bg_graphEdges, strut_edge* strut_edges);
__global__ void strut_u_init(int bg_num_vertex_b, struct strut_u_vertex* vertices_u);
__global__ void get_strut_u_degree(int num_strut_edges, strut_edge* strut_edges, struct strut_u_vertex* vertices_u);
__global__ void get_strut_u_vertices(int bg_num_edges, struct b_edge* bg_graphEdges, struct strut_u_vertex* vertices_u);
__global__ void get_zero_diff_num(int bg_num_vertex_b, struct strut_u_vertex* vertices_u, int* zero_diff_edges);

__global__ void super_vertices_init(int num_strut_vertices, int* super_vertices);
__global__ void get_new_bg_vertex_b(int num_bg_vertexb, int* super_vertices, struct strut_u_vertex* vertices_u, int* new_vertex_b, int* num_newbg_vertexb);

__global__ void prefixCopy(int prefixNum, int* old_prefix, int *new_prefix);
__global__ void getPrefixSum(int* entries, int* entriesC, int d);
__global__ void get_super_vertices(int num_strut_vertices, strut_edge* strut_edges, struct strut_u_vertex* vertices_u, int* super_vertices);
__global__ void get_new_bg_edges(int num_bg_vertex_b, int* new_bg_edges, int* prefixSum, struct strut_u_vertex* vertices_u, int* super_vertices, struct b_edge* bg_graphEdges, int * max_super_vertex);

__global__ void init_smallest_edges_weights(int num_edges, int *smallest_weights, int* smallest_edges);
/* NOTES: 
    - Remember to free all malloced and cuda malloced variables 
    - Comment out debugging statements
    - Output to file 
    - find sequential algorithm that outputs result in same way
    - compare results with that
    - Time the algorithm - put in output
    - documentation
        - read piazza and term project info for documentation
    - prep for presentation
    - submit on github - ask chonyang and email garg by thursday morning
*/

// driver
int main(int argc, char** argv){
	if(argc != 2){
		printf("mst: incorrect formatting\n");
		printf("Valid input: mst.out <Input file name>\n");
		return 0;
	}

	//***** ACQUIRE INPUT GRAPH *****//
	struct graph og_graph; // input
	get_graph(&og_graph, argv[1]);

	//debugging
	printf("Graph:\n");
	printf("vertices:%d edges:%d\n",og_graph.num_vertices, og_graph.num_edges);
	for(int i = 0; i < og_graph.num_edges; i++){
		printf("index:%d - %d   %d   %d\n", i, og_graph.edges[i].v, og_graph.edges[i].u, og_graph.edges[i].weight);
	}

	//***** CREATE BIPARTITE GRAPH *****//
	struct b_graph bg_graph;
	bg_graph.num_vertex_a = og_graph.num_vertices;
	bg_graph.num_vertex_b = og_graph.num_edges;
	bg_graph.num_bipartite_edges = og_graph.num_edges * 2;

	// allocate GPU array
	hipMalloc((void**) &(bg_graph.vertices_a), bg_graph.num_vertex_a * sizeof(struct b_vertex_a));
	hipMalloc((void**) &(bg_graph.vertices_b), bg_graph.num_vertex_b * sizeof(struct b_vertex_b));
	hipMalloc((void**) &(bg_graph.edges), bg_graph.num_bipartite_edges * sizeof(struct b_edge));

	struct edge* d_og_edges = NULL;
	hipMalloc((void**) &(d_og_edges), og_graph.num_edges * sizeof(struct edge));
	hipMemcpy(d_og_edges, og_graph.edges, og_graph.num_edges*sizeof(struct edge), hipMemcpyHostToDevice);

	get_bipartite_graph<<<(og_graph.num_edges + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(og_graph.num_edges, og_graph.num_vertices, d_og_edges, bg_graph.vertices_a, bg_graph.vertices_b, bg_graph.edges);

	//    debugging
	struct b_graph debugging;
	debugging.num_vertex_a = og_graph.num_vertices;
	debugging.num_vertex_b = og_graph.num_edges;
	debugging.num_bipartite_edges = og_graph.num_edges * 2;

	debugging.vertices_a = (struct b_vertex_a*) malloc(debugging.num_vertex_a * sizeof(struct b_vertex_a));
	debugging.vertices_b = (struct b_vertex_b*) malloc(debugging.num_vertex_b * sizeof(struct b_vertex_b));
	debugging.edges = (struct b_edge*) malloc(debugging.num_bipartite_edges * sizeof(struct b_edge));

	hipMemcpy(debugging.vertices_a, bg_graph.vertices_a, debugging.num_vertex_a * sizeof(struct b_vertex_a), hipMemcpyDeviceToHost);
	hipMemcpy(debugging.vertices_b, bg_graph.vertices_b, debugging.num_vertex_b * sizeof(struct b_vertex_b), hipMemcpyDeviceToHost);
	hipMemcpy(debugging.edges, bg_graph.edges, debugging.num_bipartite_edges * sizeof(struct b_edge), hipMemcpyDeviceToHost);

	// printf("Bipartite Graph:\n");
	// printf("verticesA: %d, verticesB: %d, edges: %d\n", debugging.num_vertex_a, debugging.num_vertex_b, debugging.num_bipartite_edges);
	// for(int i = 0; i < debugging.num_bipartite_edges; i++){
	// 	printf("index: %d - %d   %d   %d   %d\n", i, debugging.edges[i].v, debugging.edges[i].u, debugging.edges[i].cv, debugging.edges[i].weight);
	// }
	free(debugging.vertices_a);
	free(debugging.vertices_b);
    free(debugging.edges);
    
	//***** SMALLEST EDGE WEIGHT EDGE FOR EACH VERTEX IN BG_GRAPH *****//
	int* smallest_weights = NULL;
	int* smallest_edges = NULL;

	
    //***** GET SOLUTION *****//
    bool* d_mst_edges = NULL;
    bool* mst_edges = NULL;
    mst_edges = (bool*) malloc(og_graph.num_edges * sizeof(bool));

    // don't malloc again for this variable
    hipMalloc((void**) &(d_mst_edges), og_graph.num_edges* sizeof(bool));
    
    mst_edges_init<<<(og_graph.num_edges + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(og_graph.num_edges, d_mst_edges);

    int * solution_size = (int*) malloc (sizeof(int));
    *solution_size = 0;
    int* d_solutionSize = NULL;
    hipMalloc((void**) &(d_solutionSize),sizeof(int));
    hipMemcpy(d_solutionSize, solution_size, sizeof(int), hipMemcpyHostToDevice);

    int* max_super_vertex = (int*) malloc (sizeof(int));
    *max_super_vertex = bg_graph.num_vertex_a;
    
    while(*solution_size <  (og_graph.num_vertices - 1)){
        // hipMalloc((void**) &(smallest_weights), bg_graph.num_vertex_a * sizeof(int));
        // hipMalloc((void**) &(smallest_edges), bg_graph.num_vertex_a * sizeof(int));
        //get_smallest_edges<<<(bg_graph.num_bipartite_edges + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(bg_graph.num_bipartite_edges, bg_graph.num_vertex_a, bg_graph.edges, smallest_weights, smallest_edges);
        hipMalloc((void**) &(smallest_weights), *max_super_vertex * sizeof(int));
        hipMalloc((void**) &(smallest_edges), *max_super_vertex * sizeof(int));
        init_smallest_edges_weights<<<(*max_super_vertex + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(*max_super_vertex, smallest_weights, smallest_edges);
        get_smallest_edges<<<(bg_graph.num_bipartite_edges + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(bg_graph.num_bipartite_edges,*max_super_vertex, bg_graph.edges, smallest_weights, smallest_edges);
    
        // debugging
        int* debug_smallest_weights = NULL;
        debug_smallest_weights = (int*) malloc(*max_super_vertex * sizeof(int));
        hipMemcpy(debug_smallest_weights, smallest_weights, *max_super_vertex * sizeof(int), hipMemcpyDeviceToHost);
    
        // for(int i = 0; i < *max_super_vertex; i++){
        //     printf("bg index of smallest weight: %d\n", debug_smallest_weights[i]);
        // }

        int* debug_smallest_edges = NULL;
        debug_smallest_edges = (int*) malloc(*max_super_vertex * sizeof(int));
        hipMemcpy(debug_smallest_edges, smallest_edges, *max_super_vertex * sizeof(int), hipMemcpyDeviceToHost);
    
        // for(int i = 0; i < *max_super_vertex; i++){
        //     printf("bg index of smallest edge: %d\n", debug_smallest_edges[i]);
        // }

        get_mst_edges<<<(*max_super_vertex + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(*max_super_vertex , smallest_edges, bg_graph.edges, d_mst_edges);
        
        get_num_mst<<<(og_graph.num_edges  + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(og_graph.num_edges , d_mst_edges, d_solutionSize);

        // debugging
        printf("MST:\n");
        hipMemcpy(mst_edges, d_mst_edges, og_graph.num_edges * sizeof(bool), hipMemcpyDeviceToHost);
        for(int i = 0; i < og_graph.num_edges; i++){
            if(mst_edges[i] == true){
                //printf("mst edges index: %d\n", i);
                printf("index: %d - %d   %d   %d\n", i, og_graph.edges[i].v, og_graph.edges[i].u, og_graph.edges[i].weight);
            }
        }
        
        hipMemcpy(solution_size, d_solutionSize,  sizeof(int), hipMemcpyDeviceToHost);
        printf("Num MST edges found: %d\n",*solution_size);
        
        if(*solution_size <  (og_graph.num_vertices - 1)){
            //***** GET STRUT *****//
            struct strut new_strut;
            new_strut.num_v = bg_graph.num_vertex_a;
            new_strut.num_u = bg_graph.num_vertex_b;
            new_strut.num_strut_edges = bg_graph.num_vertex_a;

            struct strut_edge* d_strut_edges = NULL; 
            
            hipMalloc((void**) &(d_strut_edges), new_strut.num_v * sizeof(struct strut_edge));
            get_strut_edges<<<((bg_graph.num_vertex_a) + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(bg_graph.num_vertex_a, smallest_edges, bg_graph.edges, d_strut_edges);

            // debugging
            struct strut_edge* strut_edges = (struct strut_edge* ) malloc(new_strut.num_v * sizeof(struct strut_edge));
            hipMemcpy(strut_edges, d_strut_edges, new_strut.num_v * sizeof(struct strut_edge), hipMemcpyDeviceToHost);
            // printf("STRUT EDGES:\n");
            // for(int i = 0; i < new_strut.num_v ; i++){
            //     printf("%d   %d   %d\n", strut_edges[i].v,strut_edges[i].u, strut_edges[i].cv);
            // }

            // getting strut_u
            struct strut_u_vertex* d_vertices_u = NULL;
            hipMalloc((void**) &(d_vertices_u), new_strut.num_u * sizeof(struct strut_u_vertex));
            strut_u_init<<<((new_strut.num_u) + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(new_strut.num_u, d_vertices_u);
            get_strut_u_degree<<<((new_strut.num_v) + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(new_strut.num_v, d_strut_edges, d_vertices_u);
        
            get_strut_u_vertices<<<((bg_graph.num_bipartite_edges) + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(bg_graph.num_bipartite_edges, bg_graph.edges, d_vertices_u);

            // debugging
            struct strut_u_vertex* vertices_u = (struct strut_u_vertex* ) malloc(new_strut.num_u * sizeof(struct strut_u_vertex));
            hipMemcpy(vertices_u, d_vertices_u, new_strut.num_u * sizeof(struct strut_u_vertex), hipMemcpyDeviceToHost);
            // printf("STRUT U VERTICES DEGREE:\n");
            // for(int i = 0; i < new_strut.num_u ; i++){
            //     printf("index: %d degree: %d v1: %d v2: %d\n",i, vertices_u[i].degree,  vertices_u[i].v1,  vertices_u[i].v2);
            // }

            /* ZERO DIFF */
            int* d_zero_diff_edges = NULL;
            hipMalloc((void**) &(d_zero_diff_edges), new_strut.num_u * sizeof(int));
            get_zero_diff_num<<<((new_strut.num_u) + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(new_strut.num_u, d_vertices_u, d_zero_diff_edges);

            // debugging
            int*zero_diff_edges = (int*) malloc (sizeof(int));
            hipMemcpy(zero_diff_edges, d_zero_diff_edges, sizeof(int), hipMemcpyDeviceToHost);
            // printf("zero diff edges: %d\n", *zero_diff_edges);

            // /*SUPER VERTEX*/
            int* d_super_vertices = NULL;
            hipMalloc((void**) &(d_super_vertices), bg_graph.num_vertex_a* sizeof(int));
            super_vertices_init<<<((bg_graph.num_vertex_a) + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(bg_graph.num_vertex_a, d_super_vertices);

            int* super_vertices = (int*) malloc(bg_graph.num_vertex_a* sizeof(int));
            hipMemcpy(super_vertices, d_super_vertices,bg_graph.num_vertex_a* sizeof(int), hipMemcpyDeviceToHost);

            for(int i = 0; i < new_strut.num_u ; i++){
                int super_vertex;
                if(vertices_u[i].degree > 0){ // if incident to strut edge
                    if(super_vertices[vertices_u[i].v1 - 1] < vertices_u[i].v1){
                        super_vertex = super_vertices[vertices_u[i].v1 - 1];
                        super_vertices[vertices_u[i].v1 - 1] = super_vertex;
                        super_vertices[vertices_u[i].v2 - 1] = super_vertex;
                    }
                    else{
                        super_vertex = vertices_u[i].v1;
                        super_vertices[vertices_u[i].v1 - 1] = super_vertex;
                        super_vertices[vertices_u[i].v2 - 1] = super_vertex;
                    }
                }
            }

            hipMemcpy(d_super_vertices, super_vertices,bg_graph.num_vertex_a* sizeof(int), hipMemcpyHostToDevice);

            // debugging
            // printf("Supervertices\n:");
            // for(int i = 0; i < bg_graph.num_vertex_a ; i++){
            //     printf("vertex: %d supervertex: %d\n", i+1, super_vertices[i]);
            // }

            /******** CREATING NEW BIPARTITE GRAPH **********/
            struct b_graph new_bg_graph;
            new_bg_graph.num_vertex_a = *zero_diff_edges;
            
            int* new_num_vertex_b = NULL;
            hipMalloc((void**) &(new_num_vertex_b), sizeof(int));
            int* new_vertex_b = NULL;
            hipMalloc((void**) &(new_vertex_b), bg_graph.num_vertex_b * sizeof(int));
            get_new_bg_vertex_b<<<((bg_graph.num_vertex_b) + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(bg_graph.num_vertex_b, d_super_vertices,d_vertices_u, new_vertex_b, new_num_vertex_b);

            // num_vertex_b and num_bipartite edges
            hipMemcpy(&new_bg_graph.num_vertex_b, new_num_vertex_b, sizeof(int), hipMemcpyDeviceToHost);
            new_bg_graph.num_bipartite_edges = new_bg_graph.num_vertex_b * 2;
            
            // debugging 
            int* new_vertex_b_debug = (int*)malloc( bg_graph.num_vertex_b * sizeof(int));
            hipMemcpy(new_vertex_b_debug, new_vertex_b,  bg_graph.num_vertex_b * sizeof(int), hipMemcpyDeviceToHost);
            // printf("New Bipartie edges to choose:\n");
            // for(int i =0 ; i < bg_graph.num_vertex_b ; i++){
            //     printf("index: %d value: %d\n", i, new_vertex_b_debug[i]);
            // }
            // printf("New vertex b num: %d\n", new_bg_graph.num_vertex_b);

            int* prefixSum = NULL;
            hipMalloc((void**) &(prefixSum), bg_graph.num_vertex_b * sizeof(int));
            prefixCopy<<<((bg_graph.num_vertex_b) + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(bg_graph.num_vertex_b, new_vertex_b, prefixSum);

            //get index of bipartite edges
            int* d_prefix_helper = NULL;
            hipMalloc((void**) &(d_prefix_helper), bg_graph.num_vertex_b * sizeof(int));
            /* prefix sum belloch scan */
            int d = 1;
            while(d<bg_graph.num_vertex_b){
                getPrefixSum<<<(bg_graph.num_vertex_b + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(prefixSum, d_prefix_helper, d);
                d = 2*d;    
            }

            //debugging
            // printf("prefix sum:\n");
            // int* vertex_b_print = (int*) malloc( bg_graph.num_vertex_b * sizeof(int));
            // hipMemcpy(vertex_b_print, prefixSum,  bg_graph.num_vertex_b * sizeof(int), hipMemcpyDeviceToHost);
            // for(int i = 0; i <  bg_graph.num_vertex_b ; i++){
            //     printf("vertex: %d index: %d\n", i, vertex_b_print[i]);
            // }

            hipMalloc((void**) &(new_bg_graph.edges), new_bg_graph.num_bipartite_edges * sizeof(struct b_edge));

            int* d_max_super_vertex = NULL;
            hipMalloc((void**) &(d_max_super_vertex),sizeof(int));
            get_new_bg_edges<<<(bg_graph.num_vertex_b + THREADSPERBLOCK-1)/THREADSPERBLOCK, THREADSPERBLOCK>>>(bg_graph.num_vertex_b , new_vertex_b, prefixSum, d_vertices_u, d_super_vertices, new_bg_graph.edges, d_max_super_vertex);
            hipMemcpy(max_super_vertex, d_max_super_vertex, sizeof(int), hipMemcpyDeviceToHost);

            // debugging
            debugging.num_vertex_a = new_bg_graph.num_vertex_a;
            debugging.num_vertex_b = new_bg_graph.num_vertex_b;
            debugging.num_bipartite_edges = new_bg_graph.num_bipartite_edges ;
            
            debugging.edges = (struct b_edge*) malloc(new_bg_graph.num_bipartite_edges * sizeof(struct b_edge));
            hipMemcpy(debugging.edges, new_bg_graph.edges, new_bg_graph.num_bipartite_edges * sizeof(struct b_edge), hipMemcpyDeviceToHost);

            // printf("New Bipartite Graph:\n");
            // printf("verticesA: %d, verticesB: %d, edges: %d\n", debugging.num_vertex_a, debugging.num_vertex_b, debugging.num_bipartite_edges);
            // for(int i = 0; i < debugging.num_bipartite_edges; i++){
            //     printf("index: %d - %d   %d   %d   %d\n", i, debugging.edges[i].v, debugging.edges[i].u, debugging.edges[i].cv, debugging.edges[i].weight);
            // }

            bg_graph.num_vertex_a = new_bg_graph.num_vertex_a;
            bg_graph.num_vertex_b = new_bg_graph.num_vertex_b;
            bg_graph.num_bipartite_edges = new_bg_graph.num_bipartite_edges;

            
            hipFree(bg_graph.edges);
            hipMalloc((void**) &(bg_graph.edges), bg_graph.num_bipartite_edges * sizeof(struct b_edge));
            hipMemcpy(bg_graph.edges, debugging.edges, bg_graph.num_bipartite_edges * sizeof(struct b_edge), hipMemcpyHostToDevice);

            free(debug_smallest_weights);
            free(debug_smallest_edges);
            free(strut_edges);
            free(vertices_u);
            free(zero_diff_edges);
            free(super_vertices);
            free(new_vertex_b_debug);
            free(vertex_b_print);
            
            hipFree(d_prefix_helper);
            hipFree(prefixSum);
            hipFree(new_vertex_b);
            hipFree(new_num_vertex_b);
            hipFree(d_max_super_vertex);
            hipFree(d_super_vertices);
            hipFree(d_zero_diff_edges);
            hipFree(d_vertices_u);
            hipFree(new_num_vertex_b);
            hipFree(new_vertex_b);
            hipFree(d_strut_edges);
            hipFree(smallest_weights);
            hipFree(smallest_edges);
        }
    }

    /*end of while loop*/


    // malloc frees
    free(max_super_vertex);
    free(og_graph.edges);
    free(mst_edges);
    free(solution_size);
    free(max_super_vertex);
    

    // cuda malloc frees
    hipFree(d_solutionSize);
    hipFree(d_mst_edges);
    hipFree(mst_edges);
	hipFree(d_og_edges);
	hipFree(bg_graph.vertices_a);
	hipFree(bg_graph.vertices_b);
	hipFree(bg_graph.edges);
}

void get_graph(struct graph* og_graph, char* input){
	FILE *file;
	char buff[255];
	int num_vertices;
	int num_edges;

	file = fopen(input , "r");
    if(file == NULL){
        perror(input);
        exit(1);
    }
    else{
    	fscanf(file, "%s", buff);
    	num_vertices = atoi(buff);

    	fscanf(file, "%s", buff);
    	num_edges = atoi(buff);

    	(*og_graph).num_edges = num_edges;
    	(*og_graph).num_vertices = num_vertices;
    	(*og_graph).edges = (struct edge*) malloc(sizeof(struct edge) * num_edges);

    	for(int i = 0; i < num_edges; i++){
			fscanf(file, "%s", buff);
			(*og_graph).edges[i].v = atoi(buff);

			fscanf(file, "%s", buff);
			(*og_graph).edges[i].u = atoi(buff);

			fscanf(file, "%s", buff);
			(*og_graph).edges[i].weight = atoi(buff);
    	}
    }
    fclose(file);
}


__global__ void get_bipartite_graph(int num_edges, int num_vertices, struct edge* graphEdges, struct b_vertex_a* vertices_a, struct b_vertex_b* vertices_b, struct b_edge* bg_graphEdges) {
    int edge = threadIdx.x + blockIdx.x * blockDim.x;

    if(edge < num_edges){
    	// acquire two bipartite edges for each orginal graph edge
    	bg_graphEdges[2*edge].v = graphEdges[edge].v;
    	bg_graphEdges[2*edge].u = edge;
    	bg_graphEdges[2*edge].cv = 2*edge+1; // corresponding edge/vertex
    	bg_graphEdges[2*edge].weight = graphEdges[edge].weight;

    	bg_graphEdges[2*edge+1].v = graphEdges[edge].u;
    	bg_graphEdges[2*edge+1].u = edge;
    	bg_graphEdges[2*edge+1].cv = 2*edge; // corresponding edge/vertex
    	bg_graphEdges[2*edge+1].weight = graphEdges[edge].weight;

    	vertices_b[edge].e = edge;
    	if(edge < num_vertices)
    		vertices_a[edge].v = edge;
    }
}

__global__ void init_smallest_edges_weights(int num_edges, int *smallest_weights, int* smallest_edges){
    int edge = threadIdx.x + blockIdx.x * blockDim.x;
    if(edge < num_edges){
        smallest_weights[edge] = -1;
        smallest_edges[edge] = -1;
    }
}

// fills in smallest edges array with the index of smallest bipartite edges for each vertex (index of smallest_edges corresponds to vertex number) in graph
__global__ void get_smallest_edges(int bp_num_edges, int num_smallest_edges, struct b_edge* bg_graphEdges, int* smallest_weights, int* smallest_edges){
    int edge = threadIdx.x + blockIdx.x * blockDim.x;
    if(edge< bp_num_edges){
        int index = bg_graphEdges[edge].v - 1;
        // smallest_weights[index] = bg_graphEdges[edge].weight; // filler weight to compare with
        smallest_weights[index] = INT_MAX;
        __syncthreads(); // acquire all smallest weights
        atomicMin(&(smallest_weights[index]), bg_graphEdges[edge].weight); // save actual smallest weight

        __syncthreads(); // acquire all smallest weights

        smallest_edges[index] = bp_num_edges - 1; // filler edge number to comapre with, max edge
        // if(edge < num_smallest_edges){
        //     if(edge != index){
        //         smallest_weights[index] = -1;
        //         smallest_edges[index] = -1;
        //     }
        // }
        __syncthreads(); // acquire all smallest edges

        if(bg_graphEdges[edge].weight == smallest_weights[index]) // save smallest edge if the the bg edge has same weight as smallest weight
			//atomicMin(&(smallest_edges[index]), bg_graphEdges[edge].u);
			atomicMin(&(smallest_edges[index]), edge);
    }
}

// flags all edges to false
__global__ void mst_edges_init(int og_num_edges, bool *mst_edges){
    int edge = threadIdx.x + blockIdx.x * blockDim.x;
    if(edge < og_num_edges){
        mst_edges[edge] = false;
    }
}

// sets which edges go in mst
__global__ void get_mst_edges(int num_smallest_edges, int* smallest_edges, struct b_edge* bg_graphEdges, bool *mst_edges){
    int edge = threadIdx.x + blockIdx.x * blockDim.x;
    int bg_index;
    int vertex;
    if(edge < num_smallest_edges){
        bg_index = smallest_edges[edge];
        if(bg_index != -1){
            vertex = bg_graphEdges[bg_index].u;
            mst_edges[vertex] = true;
        }
    }
}

// gets num of mst edges in solution set
__global__ void get_num_mst(int og_num_edges, bool *mst_edges, int* num_mst){
    int edge = threadIdx.x + blockIdx.x * blockDim.x;
    if(edge < og_num_edges){
        *num_mst = 0; // reset
        __syncthreads();

        if(mst_edges[edge] == true)
            atomicAdd(num_mst, 1);
    }
}

// makes the strut edges
__global__ void get_strut_edges(int bg_num_vertices, int* smallest_edges, struct b_edge* bg_graphEdges, strut_edge* strut_edges){
    int bg_vertex = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(bg_vertex < bg_num_vertices){
        strut_edges[bg_vertex].v = bg_vertex + 1; // vertex
        strut_edges[bg_vertex].u = bg_graphEdges[smallest_edges[bg_vertex]].u; // edge index (u vertex)
        strut_edges[bg_vertex].cv = bg_graphEdges[bg_graphEdges[smallest_edges[bg_vertex]].cv].v; // save vertex that is connected to same edge index (u vertex);
    }
}

// init strut u vertices degree
__global__ void strut_u_init(int bg_num_vertex_b, struct strut_u_vertex* vertices_u){
    int vertex_b = threadIdx.x + blockIdx.x * blockDim.x;
    if(vertex_b < bg_num_vertex_b)
        vertices_u[vertex_b].degree = 0;
}


// fill in degree of strut u vertices
__global__ void get_strut_u_degree(int num_strut_vertices, strut_edge* strut_edges, struct strut_u_vertex* vertices_u){
    int strut_edge = threadIdx.x + blockIdx.x * blockDim.x;

    if(strut_edge < num_strut_vertices){
        atomicAdd(&(vertices_u[strut_edges[strut_edge].u]).degree, 1);
    }
}

// fill in what vertices the vertices_u from the strut is connected
__global__ void get_strut_u_vertices(int bg_num_edges, struct b_edge* bg_graphEdges, struct strut_u_vertex* vertices_u){
    int bg_edge = threadIdx.x + blockIdx.x * blockDim.x;
    if(bg_edge < bg_num_edges){
        if(bg_edge%2 == 0){ // only even edges
            vertices_u[bg_graphEdges[bg_edge].u].v1 = bg_graphEdges[bg_edge].v;
            vertices_u[bg_graphEdges[bg_edge].u].v2 = bg_graphEdges[bg_graphEdges[bg_edge].cv].v;
            vertices_u[bg_graphEdges[bg_edge].u].weight =  bg_graphEdges[bg_edge].weight;
        }
    }

}

// get number of zero difference vertrices u in strut
__global__ void get_zero_diff_num(int bg_num_vertex_b, struct strut_u_vertex* vertices_u, int* zero_diff_edges){
    int vertex_b = threadIdx.x + blockIdx.x * blockDim.x;
    if(vertex_b < bg_num_vertex_b){
        if(vertices_u[vertex_b].degree == 2)
            atomicAdd(zero_diff_edges, 1);
    }
}

// initialize super vertices
__global__ void super_vertices_init(int num_strut_vertices, int* super_vertices){
    int vertex = threadIdx.x + blockIdx.x * blockDim.x; 
    if(vertex < num_strut_vertices){
        super_vertices[vertex] = vertex + 1;
    }
}

// set which verticies_u will be in new bipartitie graph and get how many there are 
__global__ void get_new_bg_vertex_b(int num_bg_vertexb, int* super_vertices, struct strut_u_vertex* vertices_u, int* new_vertex_b, int* num_newbg_vertexb){
    int vertex = threadIdx.x + blockIdx.x * blockDim.x; 
    if(vertex < num_bg_vertexb){
        new_vertex_b[vertex] = 0; // setting all to false
        __syncthreads();
        if(super_vertices[vertices_u[vertex].v1 - 1] != super_vertices[vertices_u[vertex].v2 - 1]){
            new_vertex_b[vertex] = 1; 
        }
        *num_newbg_vertexb = 0;
        __syncthreads();
        if(new_vertex_b[vertex] == 1)
            atomicAdd(num_newbg_vertexb, 1);
    }
}

// copy maker
__global__ void prefixCopy(int prefixNum, int* old_prefix, int *new_prefix){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < prefixNum){
        new_prefix[index] = old_prefix[index];
    }
}
__global__ void getPrefixSum(int* entries, int* entriesC, int d) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index >= d)
        entriesC[index] = entries[index - d];
    else
        entriesC[index] = 0;

    __syncthreads();

    entries[index] = entries[index] + entriesC[index];
}

// makes new bipartite edges
__global__ void get_new_bg_edges(int num_bg_vertex_b, int* new_bg_edges, int* prefixSum, struct strut_u_vertex* vertices_u, int* super_vertices, struct b_edge* bg_graphEdges, int * max_super_vertex){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int edge1;
    int edge2;

    if(index < num_bg_vertex_b){
        if(new_bg_edges[index] == 1){
            edge1 = (prefixSum[index] - 1) * 2;
            edge2 = edge1+1;

            bg_graphEdges[edge1].v = super_vertices[vertices_u[index].v1-1];
            bg_graphEdges[edge1].u = index;
            bg_graphEdges[edge1].cv = edge2;
            bg_graphEdges[edge1].weight = vertices_u[index].weight;

            bg_graphEdges[edge2].v = super_vertices[vertices_u[index].v2-1];
            bg_graphEdges[edge2].u = index;
            bg_graphEdges[edge2].cv = edge1;
            bg_graphEdges[edge2].weight = vertices_u[index].weight;

            atomicMax(max_super_vertex, super_vertices[vertices_u[index].v1-1]);
            atomicMax(max_super_vertex, super_vertices[vertices_u[index].v2-1]);
        }
    }
}

// get what vertex each vertex is compacted to during compression of bipartite graph
__global__ void get_super_vertices(int num_strut_vertices, strut_edge* strut_edges, struct strut_u_vertex* vertices_u, int* super_vertices){
	int strut_edge = threadIdx.x + blockIdx.x * blockDim.x; 
    int cv;
    int min_cv;
    if(strut_edge < num_strut_vertices){
        min_cv = strut_edges[strut_edge].v;
        // cv = strut_edges[strut_edge].cv;
        // while(cv < min_cv){
		// 	min_cv = cv;
        //     cv = strut_edges[cv-1].cv;
        // }
        if(vertices_u[strut_edges[strut_edge].u].v1 == min_cv) 
            cv = vertices_u[strut_edges[strut_edge].u].v2;
        else
            cv = vertices_u[strut_edges[strut_edge].u].v1;
        while(cv < min_cv){
            min_cv = cv;
            if(vertices_u[strut_edges[strut_edge].u].v1 == min_cv)
                cv = vertices_u[strut_edges[cv-1].u].v2;
            else
                cv = vertices_u[strut_edges[cv-1].u].v1;
        }
        super_vertices[strut_edge] = min_cv;
    }
}